// Matrix multiplication benchmark using CUDA+CUBLAS

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
//#include <hip/hip_runtime.h>
#include <hipblas.h>

#define N 1200

#define IDX2C(i,j,ld) (((j)*(ld))+(i))

float* gen_mat(int m, int n, float start, float inc)
{
    float *res = malloc(m * n * sizeof(float));
    float acc = start;

    if (res == NULL)
        return NULL;

    for (int j = 0; j <= n; ++j) {
        for (int i = 0; i <= m; ++i) {
            res[IDX2C(i, j, m)] = acc;
            acc += inc;
        }
    }

    return res;
}

void bench_matrix_mul(int n)
{
    float *b, *devPtrB;
    float *a, *devPtrA;
    float *devPtrC;
    float alpha = 1.0f;
    float beta  = 0.0f;

    a = gen_mat(n, n, 0.0, 0.5);
    if (a == NULL) {
        printf("host memory allocation failed");
        return EXIT_FAILURE;
    }

    b = gen_mat(n, n, 4.25, 0.25);
    if (b == NULL) {
        printf("host memory allocation failed");
        return EXIT_FAILURE;
    }

    cudaStat = hipMalloc ((void**)&devPtrA, n * n * sizeof(*a));
    if (cudaStat != hipSuccess) {
        printf("device memory allocation failed for A");
        return EXIT_FAILURE;
    }

    cudaStat = hipMalloc ((void**)&devPtrB, n * n * sizeof(*b));
    if (cudaStat != hipSuccess) {
        printf("device memory allocation failed for B");
        return EXIT_FAILURE;
    }

    cudaStat = hipMalloc ((void**)&devPtrC, n * n * sizeof(float));
    if (cudaStat != hipSuccess) {
        printf("device memory allocation failed for C");
        return EXIT_FAILURE;
    }

    stat = hipblasSetMatrix (n, n, sizeof(*a), a, n, devPtrA, n);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf("data download failed for matrix A");
        hipFree (devPtrA);
        hipFree (devPtrB);
        hipFree (devPtrC);
        hipblasDestroy(handle);
        return EXIT_FAILURE;
    }

    stat = hipblasSetMatrix (n, n, sizeof(*b), b, n, devPtrB, n);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf("data download failed for matrix B");
        hipFree (devPtrA);
        hipFree (devPtrB);
        hipFree (devPtrC);
        hipblasDestroy(handle);
        return EXIT_FAILURE;
    }

    // measure time
    stat = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, 
                       &alpha, devPtrA, n, devPtrB, n, &beta, devPtrC, n);
    // measure time
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf("Error with hipblasSgemm");
    }

    // getmatrix?

    free(a);
    free(b);
    hipFree (devPtrA);
    hipFree (devPtrB);
    hipFree (devPtrC);
}

int main (void)
{
    hipError_t cudaStat;
    hipblasStatus_t stat;
    hipblasHandle_t handle;
    int i, j;
    float *b, *devPtrB;
    float *a, *devPtrA;

    if (hipSetDevice(0) != hipSuccess) {
        fprintf(stderr, "Failed to set CUDA device\n");
        return EXIT_FAILURE;
    }

    hipDeviceSynchronize();
    hipDeviceSynchronize();

    stat = hipblasCreate(&handle);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("CUBLAS initialization failed\n");
        return EXIT_FAILURE;
    }

    bench_matrix_mul(1200);

    hipblasDestroy(handle);

    return EXIT_SUCCESS;
}
